#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
int main() {
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234);
	float *hostData, *devData;
	hostData = (float *)malloc(2*sizeof(float));
	hipMalloc(&devData, 2*sizeof(float));
	hiprandGenerateUniform(gen, devData, 2);
	hipMemcpy(hostData, devData, 2*sizeof(float), hipMemcpyDeviceToHost);
	printf("ran1 = %e\nran2 = %e\n", hostData[0], hostData[1]);
	hiprandDestroyGenerator(gen); hipFree(devData); free(hostData);
}
