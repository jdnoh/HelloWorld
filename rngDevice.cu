#include <random>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void initPRNG(int seed, hiprandState *rngState)
{
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    hiprand_init(seed, tid, 0, &rngState[tid]);
}

__global__ void generate_uniform(float *data, hiprandState *rngState)
{
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    hiprandState localState = rngState[tid];
    data[tid] = hiprand_uniform(&localState);
    rngState[tid] = localState;
}

int main()
{
    int n=1<<24;
    int nThreads = 128, nBlocks = n / nThreads;

    std::random_device rd;
    unsigned int seed = rd();
    // unsigned int seed = time(0);
    printf("seed = %u\n", seed);
    
    // default PRNG 
    hiprandState *rngState_dev;
    hipMalloc(&rngState_dev, sizeof(hiprandState)*nBlocks*nThreads);
    initPRNG<<<nBlocks, nThreads>>>(seed, rngState_dev);

    float *hostData, *devData;
    size_t memSize=sizeof(float)*n;
    // host and device memory allocation
    hostData = (float *)malloc(memSize);
    hipMalloc(&devData, memSize);

    generate_uniform<<<nBlocks, nThreads>>>(devData, rngState_dev);
    
    hipMemcpy(hostData, devData, memSize, hipMemcpyDeviceToHost);

    for(int i=0; i<10; i++) printf("%d %e\n", i, hostData[i]);
    
    hipFree(rngState_dev); 
    hipFree(devData); free(hostData);
}

