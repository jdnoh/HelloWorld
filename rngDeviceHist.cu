#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

__global__ void initPRNG(int seed, hiprandState *rngState)
{
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    hiprand_init(seed, tid, 0, &rngState[tid]);
}

__global__ void generate_uniform_int(int n, unsigned int *data, int q, hiprandState *rngState)
{
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned nGrid = blockDim.x*gridDim.x;
    hiprandState localState = rngState[tid];

    for(int i=tid; i<n; i+= nGrid) 
        data[i] = hiprand(&localState)%q;

    rngState[tid] = localState;
}

int main()
{

    int nBlocks = 128, nThreads = 128;
    int seed = 1234;
    int q=4;
    int hist[q];
    
    // default PRNG 
    hiprandState *rngState_dev;
    hipMalloc(&rngState_dev, sizeof(hiprandState)*nBlocks*nThreads);
    initPRNG<<<nBlocks, nThreads>>>(seed, rngState_dev);

    int   n=1<<28;
    unsigned int *hostData, *devData;
    size_t memSize=sizeof(unsigned int)*n;
    // host and device memory allocation
    hostData = (unsigned int *)malloc(memSize);
    hipMalloc(&devData, memSize);

    for(int i=0; i<10; i++) 
        generate_uniform_int<<<nBlocks, nThreads>>>(n, devData, q, rngState_dev);
    
    hipMemcpy(hostData, devData, memSize, hipMemcpyDeviceToHost);

    // histogram
    for(int i=0; i<q; i++) hist[i] = 0;
    for(int i=0; i<n; i++) {
        hist[hostData[i]] ++;
    }
    for(int i=0; i<q; i++) printf("%d %d\n", i, hist[i]);

    thrust::sort(thrust::device_ptr<unsigned int>(devData),
            thrust::device_ptr<unsigned int>(devData)+n);

    thrust::device_vector<unsigned int> histogram(q,0);
    thrust::counting_iterator<unsigned int> search_begin(0);
    thrust::upper_bound(thrust::device_ptr<unsigned int>(devData),
            thrust::device_ptr<unsigned int>(devData)+n,
            search_begin, search_begin+q, histogram.begin());
    thrust::adjacent_difference(histogram.begin(), histogram.end(),
            histogram.begin());

    for(int i=0; i<histogram.size(); i++)
        std::cout << i << " " << histogram[i] << "\n";
    
    hipFree(rngState_dev); 
    hipFree(devData); free(hostData);
}

