#include "hip/hip_runtime.h"
#include <iostream>

// Kernel function to add the elements of two arrays
__global__ void add(int n, int *x, int *y, int a)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid<n) x[tid] = a*x[tid] + y[tid];
}

int main(void)
{
    int dNum = 1<<24;
    int *x, *y;
    // memory size for each array
    size_t memSize = sizeof(int)*dNum;

    // unified memory allocation
    hipMallocManaged(&x, memSize); 
    hipMallocManaged(&y, memSize);

    // initialization in CPU
    for(int i=0; i<dNum; i++) { x[i] = 1; y[i] = 2; }

    // device id
    int gpuId;
    hipGetDevice(&gpuId);

    // pre-fetch 'x' and 'y' to the device
    hipMemPrefetchAsync(x, memSize, gpuId);
    hipMemPrefetchAsync(y, memSize, gpuId);

    // grid configuration (dNum = numBlocks * numThreads)
    int numThreads = 512;
    int numBlocks  = (dNum + numThreads-1) / numThreads;

    // Run kernel on the device
    add<<<numBlocks, numThreads>>>(dNum, x, y, 2);
    // explicit barrier
    hipDeviceSynchronize();

    hipMemPrefetchAsync(x, memSize, hipCpuDeviceId);
    hipMemPrefetchAsync(y, memSize, hipCpuDeviceId);
    for(int i=0; i<dNum; i++) if(x[i]!=4) printf("Error!\n");

    // Free memory
    hipFree(x); hipFree(y);
    return 0;
}
