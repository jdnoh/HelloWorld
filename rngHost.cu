#include <random>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <time.h>

int main()
{
    
    hiprandGenerator_t gen;
    // default (WOWXOR) or Mersenne-Trister pseudo random number generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); 
    // curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_MT19937); 

    // initialize the PRNG with seed
    // std::random_device rd;
    // unsigned int seed = rd();
    unsigned int seed = time(0);
    printf("seed = %u\n", seed);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);

    float *hostData, *devData;
    int   n=1<<24;
    size_t memSize=sizeof(float)*n;
    // host and device memory allocation
    hostData = (float *)malloc(memSize);
    hipMalloc(&devData, memSize);

    // generate n random numbers in (0,1] on the device array 
    hiprandGenerateUniform(gen, devData, n);

    hipMemcpy(hostData, devData, memSize, hipMemcpyDeviceToHost);

    for(int i=0; i<10; i++) printf("%d %e\n", i, hostData[i]);

    hiprandDestroyGenerator(gen);
    hipFree(devData); free(hostData);
}

